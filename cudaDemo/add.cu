
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

int add(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

int main(void) 
{
    int N = 1<<20;
    float *x = new float[N];
    float *y = new float[N];

    //init x, y arrs on host
    for (int i = 0; i < N ; i++ )
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on GPU 
    add(N,x,y);

    // Free mem 
    delete [] x;
    delete [] y;

    return 0;
}